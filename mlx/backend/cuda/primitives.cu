// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/dtype_utils.cuh"
#include "mlx/backend/cuda/kernels/arange.cuh"
#include "mlx/backend/cuda/kernels/arg_reduce.cuh"
#include "mlx/backend/cuda/kernels/fp16_math.cuh"
#include "mlx/backend/cuda/kernels/random.cuh"
#include "mlx/distributed/primitives.h"
#include "mlx/dtype_utils.h"
#include "mlx/fast_primitives.h"
#include "mlx/primitives.h"

#include <nvtx3/nvtx3.hpp>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>

#include <cassert>

namespace mlx::core {

void Arange::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("Arange::eval_gpu");
  assert(inputs.size() == 0);
  out.set_data(allocator::malloc(out.nbytes()));
  if (out.size() == 0) {
    return;
  }
  auto& s = stream();
  auto& encoder = cu::get_command_encoder(s);
  encoder.set_output_array(out);
  encoder.launch_kernel([&, this](hipStream_t stream) {
    MLX_SWITCH_INT_FLOAT_TYPES_CHECKED(out.dtype(), "Arange", CTYPE, {
      using OutType = cuda_type_t<CTYPE>;
      CTYPE step =
          static_cast<CTYPE>(start_ + step_) - static_cast<CTYPE>(start_);
      thrust::transform(
          cu::thrust_policy(stream),
          thrust::counting_iterator<uint32_t>(0),
          thrust::counting_iterator<uint32_t>(out.data_size()),
          thrust::device_pointer_cast(out.data<OutType>()),
          cu::Arange<OutType>{
              static_cast<OutType>(start_), static_cast<OutType>(step)});
    });
  });
}

void ArgReduce::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("ArgReduce::eval_gpu");
  assert(inputs.size() == 1);
  auto& in = inputs[0];
  out.set_data(allocator::malloc(out.nbytes()));
  auto& s = stream();

  // Prepare the shapes, strides and axis arguments.
  auto in_strides = in.strides();
  auto shape = in.shape();
  auto out_strides = out.strides();
  auto axis_stride = in_strides[axis_];
  size_t axis_size = shape[axis_];
  if (out_strides.size() == in_strides.size()) {
    out_strides.erase(out_strides.begin() + axis_);
  }
  in_strides.erase(in_strides.begin() + axis_);
  shape.erase(shape.begin() + axis_);
  size_t ndim = shape.size();

  // ArgReduce.
  auto& encoder = cu::get_command_encoder(s);
  encoder.set_input_array(in);
  encoder.set_output_array(out);
  encoder.launch_kernel([&](hipStream_t stream) {
    MLX_SWITCH_REAL_TYPES_CHECKED(in.dtype(), "ArgReduce", CTYPE, {
      using InType = cuda_type_t<CTYPE>;
      constexpr uint32_t N_READS = 4;
      MLX_SWITCH_BLOCK_DIM(cuda::ceil_div(axis_size, N_READS), BLOCK_DIM, {
        auto kernel = &cu::arg_reduce_general<
            InType,
            cu::ArgMax<InType>,
            BLOCK_DIM,
            N_READS>;
        if (reduce_type_ == ArgReduce::ArgMin) {
          kernel = &cu::arg_reduce_general<
              InType,
              cu::ArgMin<InType>,
              BLOCK_DIM,
              N_READS>;
        }
        kernel<<<out.data_size(), BLOCK_DIM, 0, stream>>>(
            in.data<InType>(),
            out.data<uint32_t>(),
            cu::const_param(shape),
            cu::const_param(in_strides),
            cu::const_param(out_strides),
            ndim,
            axis_stride,
            axis_size);
      });
    });
  });
}

void RandomBits::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("RandomBits::eval_gpu");
  assert(inputs.size() == 1);

  // keys has shape (N1, ..., NK, 2)
  // out has shape (N1, ..., NK, M1, M2, ...)
  auto& keys = inputs[0];
  size_t num_keys = keys.size() / 2;

  size_t elems_per_key = out.size() / num_keys;
  size_t bytes_per_key = out.itemsize() * elems_per_key;
  out.set_data(allocator::malloc(out.nbytes()));
  if (out.size() == 0) {
    return;
  }

  size_t out_per_key = (bytes_per_key + 4 - 1) / 4;
  size_t half_size = out_per_key / 2;
  bool odd = out_per_key % 2;

  auto& s = stream();
  auto& encoder = cu::get_command_encoder(s);
  encoder.set_input_array(keys);
  encoder.set_output_array(out);
  encoder.launch_kernel([&](hipStream_t stream) {
    dim3 grid_dim{
        static_cast<uint32_t>(num_keys),
        static_cast<uint32_t>(half_size + odd)};
    dim3 block_dim = get_block_dims(grid_dim.x, grid_dim.y, 1);
    dim3 num_blocks{
        cuda::ceil_div(grid_dim.x, block_dim.x),
        cuda::ceil_div(grid_dim.y, block_dim.y)};
    if (keys.flags().row_contiguous) {
      cu::rbitsc<<<num_blocks, block_dim, 0, stream>>>(
          keys.data<uint32_t>(),
          out.data<uint8_t>(),
          grid_dim,
          odd,
          bytes_per_key);
    } else {
      cu::rbits<<<num_blocks, block_dim, 0, stream>>>(
          keys.data<uint32_t>(),
          out.data<uint8_t>(),
          grid_dim,
          odd,
          bytes_per_key,
          keys.ndim(),
          cu::const_param(keys.shape()),
          cu::const_param(keys.strides()));
    }
  });
}

#define NO_GPU_MULTI(func)                                             \
  void func::eval_gpu(                                                 \
      const std::vector<array>& inputs, std::vector<array>& outputs) { \
    throw std::runtime_error(#func " has no CUDA implementation.");    \
  }

#define NO_GPU(func)                                                  \
  void func::eval_gpu(const std::vector<array>& inputs, array& out) { \
    throw std::runtime_error(#func " has no CUDA implementation.");   \
  }

NO_GPU(AddMM)
NO_GPU(ArgPartition)
NO_GPU(ArgSort)
NO_GPU(BlockMaskedMM)
NO_GPU_MULTI(Compiled)
NO_GPU(Convolution)
NO_GPU_MULTI(DivMod)
NO_GPU(DynamicSlice)
NO_GPU(DynamicSliceUpdate)
NO_GPU(FFT)
NO_GPU(Gather)
NO_GPU(GatherAxis)
NO_GPU(GatherMM)
NO_GPU(GatherQMM)
NO_GPU(Hadamard)
NO_GPU(Load)
NO_GPU(LogSumExp)
NO_GPU_MULTI(LUF)
NO_GPU(Matmul)
NO_GPU(Partition)
NO_GPU_MULTI(QRF)
NO_GPU(QuantizedMatmul)
NO_GPU(Scan)
NO_GPU(Scatter)
NO_GPU(ScatterAxis)
NO_GPU(Select)
NO_GPU(SliceUpdate)
NO_GPU(Softmax)
NO_GPU(Sort)
NO_GPU_MULTI(SVD)
NO_GPU(Inverse)
NO_GPU(Cholesky)
NO_GPU_MULTI(Eigh)

namespace fast {
NO_GPU_MULTI(LayerNorm)
NO_GPU_MULTI(LayerNormVJP)
NO_GPU_MULTI(RMSNorm)
NO_GPU_MULTI(RMSNormVJP)
NO_GPU_MULTI(RoPE)
NO_GPU(ScaledDotProductAttention)
NO_GPU_MULTI(AffineQuantize)
NO_GPU_MULTI(CustomKernel)
} // namespace fast

namespace distributed {
NO_GPU_MULTI(AllReduce)
NO_GPU_MULTI(AllGather)
NO_GPU_MULTI(Send)
NO_GPU_MULTI(Recv)
} // namespace distributed

} // namespace mlx::core
